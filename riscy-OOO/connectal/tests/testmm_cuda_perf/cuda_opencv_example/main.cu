#include "hip/hip_runtime.h"
/* Copyright (c) 2014 Quanta Research Cambridge, Inc
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included
 * in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS
 * OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */
#include<iostream>
#include<cstdio>
#include<opencv2/core/core.hpp>
#include<opencv2/highgui/highgui.hpp>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>

using std::cout;
using std::endl;

static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
	if(err!=hipSuccess)
	{
		fprintf(stderr,"%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n",msg,file_name,line_number,hipGetErrorString(err));
		std::cin.get();
		exit(EXIT_FAILURE);
	}
}

#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)


__global__ void bgr_to_gray_kernel( unsigned char* input, 
				    unsigned char* output, 
				    int width,
				    int height,
				    int colorWidthStep,
				    int grayWidthStep)
{
	//2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	//Only valid threads perform memory I/O
	if((xIndex<width) && (yIndex<height))
	{
		//Location of colored pixel in input
		const int color_tid = yIndex * colorWidthStep + (3 * xIndex);
		
		//Location of gray pixel in output
		const int gray_tid  = yIndex * grayWidthStep + xIndex;

		const unsigned char blue	= input[color_tid];
		const unsigned char green	= input[color_tid + 1];
		const unsigned char red		= input[color_tid + 2];

		const float gray = red * 0.3f + green * 0.59f + blue * 0.11f;

		output[gray_tid] = static_cast<unsigned char>(gray);
	}
}

void convert_to_gray(const cv::Mat& input, cv::Mat& output)
{
	//Calculate total number of bytes of input and output image
	const int colorBytes = input.step * input.rows;
	const int grayBytes = output.step * output.rows;

	unsigned char *d_input, *d_output;

	//Allocate device memory
	SAFE_CALL(hipMalloc<unsigned char>(&d_input,colorBytes),"CUDA Malloc Failed");
	SAFE_CALL(hipMalloc<unsigned char>(&d_output,grayBytes),"CUDA Malloc Failed");

	//Copy data from OpenCV input image to device memory
	SAFE_CALL(hipMemcpy(d_input,input.ptr(),colorBytes,hipMemcpyHostToDevice),"CUDA Memcpy Host To Device Failed");

	//Specify a reasonable block size
	const dim3 block(16,16);

	//Calculate grid size to cover the whole image
	const dim3 grid((input.cols + block.x - 1)/block.x, (input.rows + block.y - 1)/block.y);

	//Launch the color conversion kernel
	bgr_to_gray_kernel<<<grid,block>>>(d_input,d_output,input.cols,input.rows,input.step,output.step);

	//Synchronize to check for any kernel launch errors
	SAFE_CALL(hipDeviceSynchronize(),"Kernel Launch Failed");

	//Copy back data from destination device meory to OpenCV output image
	SAFE_CALL(hipMemcpy(output.ptr(),d_output,grayBytes,hipMemcpyDeviceToHost),"CUDA Memcpy Host To Device Failed");

	//Free the device memory
	SAFE_CALL(hipFree(d_input),"CUDA Free Failed");
	SAFE_CALL(hipFree(d_output),"CUDA Free Failed");
}

#define SIGMOID(x) ((x < -8.0) ? -8.0 : ((x > 8.0) ? 8.0 : (1 / (1 + expf(-x)))))

float sigmoid(float x)
{
  return SIGMOID(x);
}

__global__ void ssigmoid( float* input, 
			  float* output, 
			  int width,
			  int height)
{
  //2D Index of current thread
  const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
  const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
  
  //Only valid threads perform memory I/O
  if((xIndex<width) && (yIndex<height)) {
    int offs = (yIndex*width)+xIndex;
    output[offs] = SIGMOID(input[offs]);
  }
}


void map_sigmoid(const cv::Mat& input, cv::Mat& output)
{
  //Calculate total number of bytes of input and output image
  const int inputBytes = input.step * input.rows;
  const int outputBytes = output.step * output.rows;
  
  float *d_input, *d_output;
  
  //Allocate device memory
  SAFE_CALL(hipMalloc<float>(&d_input,inputBytes),"CUDA Malloc Failed");
  SAFE_CALL(hipMalloc<float>(&d_output,outputBytes),"CUDA Malloc Failed");
  
  //Copy data from OpenCV input image to device memory
  SAFE_CALL(hipMemcpy(d_input,input.ptr(),inputBytes,hipMemcpyHostToDevice),"CUDA Memcpy Host To Device Failed");
  
  //Specify a reasonable block size
  const dim3 block(16,16);
  
  //Calculate grid size to cover the whole image
  const dim3 grid((input.cols + block.x - 1)/block.x, (input.rows + block.y - 1)/block.y);
  
  //Launch the input conversion kernel
  ssigmoid<<<grid,block>>>(d_input,d_output,input.cols,input.rows);
  
  //Synchronize to check for any kernel launch errors
  SAFE_CALL(hipDeviceSynchronize(),"Kernel Launch Failed");
  
  //Copy back data from destination device meory to OpenCV output image
  SAFE_CALL(hipMemcpy(output.ptr(),d_output,outputBytes,hipMemcpyDeviceToHost),"CUDA Memcpy Host To Device Failed");
  
  //Free the device memory
  SAFE_CALL(hipFree(d_input),"CUDA Free Failed");
  SAFE_CALL(hipFree(d_output),"CUDA Free Failed");
}

